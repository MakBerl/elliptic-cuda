//menyelesaikan persamaan eliptik 2D untuk persegi (X = Y = N)

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <math.h>

void writeToFile(int X, int Y, int length, char* filename1, float* arr) {
	//print ke file
	FILE * pFile = fopen(filename1,"w");
	fprintf(pFile, "%d\n%d\n",X,Y);
	for (int i=0;i<Y;i++) {
		for(int j=0;j<X;j++) {
			fprintf(pFile,"%.2f ",arr[(i*X)+j]);
		}
		fprintf(pFile, "\n");
	}
	fclose(pFile);
}

float absol(float input) {
	if(input < 0) {
		return (-input);
	}
	else {
		return input;
	}
}

void cpuJacobi (float * input, float * output, float * error, char ad_n, char ad_e, char ad_s, char ad_w, int N, int jumlahElemen) {
	for (int i = 0; i < jumlahElemen; i++) {
		//float A = input[i], B = input[i], C = input[i], D = input[i];
		//tetap pada batas:
		//y = 0 --> i % N = 0
		//x = 0 --> i / N = 0
		//y = y --> i % N = N-1
		//x = x --> i / N = N-1
		if(i == 0 || i == N-1 || ((i / N == 0) && (i % N == N-1)) || ((i % N == 0) && (i / N == N-1))) {
			output[i] = input[i];
		}
		else if((i%N == 0) || (i/N == 0) || (i%N == N-1) || (i/N == N-1)) {
			if(i/N == 0 && ad_n) {
				output[i] = (input[i+1] + input[i+N] + input[i-1])/3;
			}
			else if(i%N == N-1 && ad_e) {
				output[i] = (input[i-1] + input[i+N] + input[i-N])/3;
			}
			else if(i/N == N-1 && ad_s) {
				output[i] = (input[i+1] + input[i-1] + input[i-N])/3;
			}
			else if(i%N == 0 && ad_w){
				output[i] = (input[i+1] + input[i+N] + input[i-N])/3;
			}
			else{
				output[i] = input[i];
			}
		}
		else {	//tidak pada batas
			//lakukan perhitungan Jacobi
			output[i] = (input[i-1] + input[i+1] + input[i-N] + input[i+N])/(4);
		}
		//hitung error
		error[i] = output[i]-input[i];

		//input[i] = output[i];
	}
	for(int i = 0; i< jumlahElemen; i++) {
		input[i] = output[i];
	}
}

void cpuPGS(float * input, float * output, float * error, char ad_n, char ad_e, char ad_s, char ad_w, int N, int jumlahElemen) {
	//float A = 0, B = 0, C = 0, D = 0;
	for (int i = 0; i < jumlahElemen; i++) {
		//tetap pada batas:
		//y = 0 --> i % N = 0
		//x = 0 --> i / N = 0
		//y = y --> i % N = N-1
		//x = x --> i / N = N-1
		if(i == 0 || i == N-1 || ((i / N == 0) && (i % N == N-1)) || ((i % N == 0) && (i / N == N-1))) {
			output[i] = input[i];
		}
		else if((i%N == 0) || (i/N == 0) || (i%N == N-1) || (i/N == N-1)) {
			if(i/N == 0 && ad_n) {
				output[i] = (input[i+1] + input[i+N] + input[i-1])/3;
			}
			else if(i%N == N-1 && ad_e) {
				output[i] = (input[i-1] + input[i+N] + input[i-N])/3;
			}
			else if(i/N == N-1 && ad_s) {
				output[i] = (input[i+1] + input[i-1] + input[i-N])/3;
			}
			else if(i%N == 0 && ad_w){
				output[i] = (input[i+1] + input[i+N] + input[i-N])/3;
			}
			else{
				output[i] = input[i];
			}
		}
		else {	//tidak pada batas
			//lakukan perhitungan PGS
			output[i] = (output[i-1] + input[i+1] + output[i-N] + input[i+N])/(4);
		}
		//hitung error
		error[i] = output[i]-input[i];

		//input[i] = output[i];
	}
	for(int i = 0; i< jumlahElemen; i++) {
		input[i] = output[i];
	}
}

void cpuPSOR(float * input, float * output, float * error, char ad_n, char ad_e, char ad_s, char ad_w, int N, int jumlahElemen, float omega) {
	//float A = 0, B = 0, C = 0, D = 0;
	for (int i = 0; i < jumlahElemen; i++) {
		//tetap pada batas:
		//y = 0 --> i % N = 0
		//x = 0 --> i / N = 0
		//y = y --> i % N = N-1
		//x = x --> i / N = N-1
		if(i == 0 || i == N-1 || ((i / N == 0) && (i % N == N-1)) || ((i % N == 0) && (i / N == N-1))) {
			output[i] = input[i];
		}
		else if((i%N == 0) || (i/N == 0) || (i%N == N-1) || (i/N == N-1)) {
			if(i/N == 0 && ad_n) {
				output[i] = (input[i+1] + input[i+N] + input[i-1])/3;
			}
			else if(i%N == N-1 && ad_e) {
				output[i] = (input[i-1] + input[i+N] + input[i-N])/3;
			}
			else if(i/N == N-1 && ad_s) {
				output[i] = (input[i+1] + input[i-1] + input[i-N])/3;
			}
			else if(i%N == 0 && ad_w){
				output[i] = (input[i+1] + input[i+N] + input[i-N])/3;
			}
			else{
				output[i] = input[i];
			}
		}
		else {	//tidak pada batas
			//lakukan perhitungan PSOR
			output[i] = input[i] + ((output[i-1] + input[i+1] + output[i-N] + input[i+N] - (input[i]*4))*omega/(4));
		}
		//hitung error
		error[i] = output[i]-input[i];

		//input[i] = output[i];
	}
	for(int i = 0; i< jumlahElemen; i++) {
		input[i] = output[i];
	}
}

__global__ void gpuJacobi (float * input, float * error, char ad_n, char ad_e, char ad_s, char ad_w, int N) {
	//float A = 0, B = 0, C = 0, D = 0;
	//thread ID
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int tidy = (blockIdx.y * blockDim.y) + threadIdx.y;
	int tid = (tidy * N) + tidx;	// This gives every thread a unique ID.
	
	if(tid<(N*N)) {
		float hasil = 0;
		//tetap pada batas:
		//y = 0 --> i % N = 0
		//x = 0 --> i / N = 0
		//y = y --> i % N = N-1
		//x = x --> i / N = N-1
		if(tid == 0 || tid == N-1 || ((tid / N == 0) && (tid % N == N-1)) || ((tid % N == 0) && (tid / N == N-1))) {
			hasil = input[tid];
		}
		else if((tid%N == 0) || (tid/N == 0) || (tid%N == N-1) || (tid/N == N-1)) {
			if(tid/N == 0 && ad_n) {
				hasil = (input[tid+1] + input[tid+N] + input[tid-1])/3;
			}
			else if(tid%N == N-1 && ad_e) {
				hasil = (input[tid-1] + input[tid+N] + input[tid-N])/3;
			}
			else if(tid/N == N-1 && ad_s) {
				hasil = (input[tid+1] + input[tid-1] + input[tid-N])/3;
			}
			else if(tid%N == 0 && ad_w){
				hasil = (input[tid+1] + input[tid+N] + input[tid-N])/3;
			}
			else{
				hasil = input[tid];
			}
		}
		else {	//tidak pada batas
			//lakukan perhitungan Jacobi
			hasil = (input[tid-1] + input[tid+1] + input[tid-N] + input[tid+N])/(4);
			error[tid] = hasil-input[tid];
		}
		input[tid] = hasil;
		//hitung error
	}
}

__global__ void gpuPGSRed (float * input, float * error, char ad_n, char ad_e, char ad_s, char ad_w, int N) {
	//float A = 0, B = 0, C = 0, D = 0
	float hasil = 0;
	//thread ID
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int tidy = (blockIdx.y * blockDim.y) + threadIdx.y;
	int tid = (tidy * N) + tidx;	// This gives every thread a unique ID.
	//int blid = (blockIdx.y * blockDim.x) + blockIdx.x;
	
	if(tid<(N*N)) {
		//tetap pada batas:
		//y = 0 --> i % N = 0
		//x = 0 --> i / N = 0
		//y = y --> i % N = N-1
		//x = x --> i / N = N-1
		if (tid%2 != (tid/N)%2) {
			if(tid == 0 || tid == N-1 || ((tid / N == 0) && (tid % N == N-1)) || ((tid % N == 0) && (tid / N == N-1))) {
				hasil = input[tid];
			}
			else if((tid%N == 0) || (tid/N == 0) || (tid%N == N-1) || (tid/N == N-1)) {
				if(tid/N == 0 && ad_n) {
					hasil = (input[tid+1] + input[tid+N] + input[tid-1])/3;
				}
				else if(tid%N == N-1 && ad_e) {
					hasil = (input[tid-1] + input[tid+N] + input[tid-N])/3;
				}
				else if(tid/N == N-1 && ad_s) {
					hasil = (input[tid+1] + input[tid-1] + input[tid-N])/3;
				}
				else if(tid%N == 0 && ad_w){
					hasil = (input[tid+1] + input[tid+N] + input[tid-N])/3;
				}
				else{
					hasil = input[tid];
				}
			}
			else {	//tidak pada batas
				//lakukan perhitungan PGS
				//if (tid%2 != (tid/N)%2) {
					hasil = (input[tid-1] + input[tid+1] + input[tid-N] + input[tid+N])/(4);
					error[tid] = hasil-input[tid];
				//}
			}
		input[tid] = hasil;
		}
		
		//hitung error
	}
}

__global__ void gpuPGSBlack (float * input, float * error, char ad_n, char ad_e, char ad_s, char ad_w, int N) {
	//float A = 0, B = 0, C = 0, D = 0;
	float hasil = 0;
	//thread ID
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int tidy = (blockIdx.y * blockDim.y) + threadIdx.y;
	int tid = (tidy * N) + tidx;	// This gives every thread a unique ID.
	//int blid = (blockIdx.y * blockDim.x) + blockIdx.x;
	
	if(tid<(N*N)) {
		//tetap pada batas:
		//y = 0 --> i % N = 0
		//x = 0 --> i / N = 0
		//y = y --> i % N = N-1
		//x = x --> i / N = N-1
		if (tid%2 == (tid/N)%2) {
			if(tid == 0 || tid == N-1 || ((tid / N == 0) && (tid % N == N-1)) || ((tid % N == 0) && (tid / N == N-1))) {
				hasil = input[tid];
			}
			else if((tid%N == 0) || (tid/N == 0) || (tid%N == N-1) || (tid/N == N-1)) {
				if(tid/N == 0 && ad_n) {
					hasil = (input[tid+1] + input[tid+N] + input[tid-1])/3;
				}
				else if(tid%N == N-1 && ad_e) {
					hasil = (input[tid-1] + input[tid+N] + input[tid-N])/3;
				}
				else if(tid/N == N-1 && ad_s) {
					hasil = (input[tid+1] + input[tid-1] + input[tid-N])/3;
				}
				else if(tid%N == 0 && ad_w){
					hasil = (input[tid+1] + input[tid+N] + input[tid-N])/3;
				}
				else{
					hasil = input[tid];
				}
			}
			else {	//tidak pada batas
				//lakukan perhitungan PGS
				//if (tid%2 != (tid/N)%2) {
					hasil = (input[tid-1] + input[tid+1] + input[tid-N] + input[tid+N])/(4);
					error[tid] = hasil-input[tid];
				//}
			}
		input[tid] = hasil;
		}
	}
}

__global__ void gpuPSORRed (float * input, float * error, char ad_n, char ad_e, char ad_s, char ad_w, int N, float omega) {
	//float A = 0, B = 0, C = 0, D = 0;
	float hasil = 0;
	//thread ID
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int tidy = (blockIdx.y * blockDim.y) + threadIdx.y;
	int tid = (tidy * N) + tidx;	// This gives every thread a unique ID.
	//int blid = (blockIdx.y * blockDim.x) + blockIdx.x;
	
	if(tid<(N*N)) {
		//tetap pada batas:
		//y = 0 --> i % N = 0
		//x = 0 --> i / N = 0
		//y = y --> i % N = N-1
		//x = x --> i / N = N-1
		if (tid%2 != (tid/N)%2) {
			if(tid == 0 || tid == N-1 || ((tid / N == 0) && (tid % N == N-1)) || ((tid % N == 0) && (tid / N == N-1))) {
				hasil = input[tid];
			}
			else if((tid%N == 0) || (tid/N == 0) || (tid%N == N-1) || (tid/N == N-1)) {
				if(tid/N == 0 && ad_n) {
					hasil = (input[tid+1] + input[tid+N] + input[tid-1])/3;
				}
				else if(tid%N == N-1 && ad_e) {
					hasil = (input[tid-1] + input[tid+N] + input[tid-N])/3;
				}
				else if(tid/N == N-1 && ad_s) {
					hasil = (input[tid+1] + input[tid-1] + input[tid-N])/3;
				}
				else if(tid%N == 0 && ad_w){
					hasil = (input[tid+1] + input[tid+N] + input[tid-N])/3;
				}
				else{
					hasil = input[tid];
				}
			}
			else {	//tidak pada batas
				//lakukan perhitungan PSOR
				//if (tid%2 != (tid/N)%2) {
					hasil = input[tid] + ((input[tid-1] + input[tid+1] + input[tid-N] + input[tid+N] - (input[tid]*4))*(omega/4));
					error[tid] = hasil-input[tid];		
				//}
			}
			//hitung error
			input[tid] = hasil;
		}
	}
}

__global__ void gpuPSORBlack (float * input, float * error, char ad_n, char ad_e, char ad_s, char ad_w, int N, float omega) {
	//float A = 0, B = 0, C = 0, D = 0;
	float hasil = 0;
	//thread ID
	int tidx = (blockIdx.x * blockDim.x) + threadIdx.x;
	int tidy = (blockIdx.y * blockDim.y) + threadIdx.y;
	int tid = (tidy * N) + tidx;	// This gives every thread a unique ID.
	//int blid = (blockIdx.y * blockDim.x) + blockIdx.x;
	
	if(tid<(N*N)) {
		//tetap pada batas:
		//y = 0 --> i % N = 0
		//x = 0 --> i / N = 0
		//y = y --> i % N = N-1
		//x = x --> i / N = N-1
		if (tid%2 == (tid/N)%2) {
			if(tid == 0 || tid == N-1 || ((tid / N == 0) && (tid % N == N-1)) || ((tid % N == 0) && (tid / N == N-1))) {
				hasil = input[tid];
			}
			else if((tid%N == 0) || (tid/N == 0) || (tid%N == N-1) || (tid/N == N-1)) {
				if(tid/N == 0 && ad_n) {
					hasil = (input[tid+1] + input[tid+N] + input[tid-1])/3;
				}
				else if(tid%N == N-1 && ad_e) {
					hasil = (input[tid-1] + input[tid+N] + input[tid-N])/3;
				}
				else if(tid/N == N-1 && ad_s) {
					hasil = (input[tid+1] + input[tid-1] + input[tid-N])/3;
				}
				else if(tid%N == 0 && ad_w){
					hasil = (input[tid+1] + input[tid+N] + input[tid-N])/3;
				}
				else{
					hasil = input[tid];
				}
			}
			else {	//tidak pada batas
				//lakukan perhitungan PSOR
					hasil = input[tid] + ((input[tid-1] + input[tid+1] + input[tid-N] + input[tid+N] - (input[tid]*4))*(omega/4));
					error[tid] = hasil-input[tid];		
			}
			//hitung error
			input[tid] = hasil;
		}
	}
}

int main (int argc, char** argv) {
	int SISI_MATRIKS = 384;
	int totalElemen = SISI_MATRIKS*SISI_MATRIKS;
	//float pi =  3.14159265;

	char ad_n = 0, ad_e = 0, ad_s = 0, ad_w = 1;
	float omega = 1.6;
	float *awal_in, *awal_out, *awal_err;
	float *host_in, *host_out, *host_err;
	float *dev_in, *dev_err;
	float total_error = -1;
	float MAX_ERROR = 0.05;
	int CPUIter = 0, CPUIter2 = 0, CPUIter3 = 0, GPUIter = 0, GPUIter2 = 0, GPUIter3 = 0;
	float CPUTime = 0, CPUTime2 = 0, CPUTime3 = 0, GPUTime = 0, GPUTime2 = 0, GPUTime3 = 0;
	dim3 jumlahBlock, threadPerBlock;
	clock_t t1;
	clock_t t2;

	//inialisasi
	awal_in = (float *)malloc(sizeof(float) * totalElemen);
	awal_out = (float *)malloc(sizeof(float) * totalElemen);
	awal_err = (float *)malloc(sizeof(float) * totalElemen);

	//instansiasi
	printf("generating domain matrix...");
	for(int i = 0; i<totalElemen; i++) {
		//kondisi batas
		if(i<SISI_MATRIKS) {
			awal_in[i] = 100;
		}
		else {
			awal_in[i] = 0;
		}
		awal_out[i] = awal_in[i];
		awal_err[i] = 0;
	}
	printf("done\n");
	
	printf("\nMulai pengujian...\n------------------------------------------\n");
	printf("CPU\n");
	//pengujian
	printf("Metode Jacobi.....");
	//---CPU Jacobi---
	//alokasi memori domain
	host_in = (float *)malloc(sizeof(float) * totalElemen);
	host_out = (float *)malloc(sizeof(float) * totalElemen);
	host_err = (float *)malloc(sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	for(int i = 0; i<totalElemen; i++) {
		host_in[i] = awal_in[i];
		host_out[i] = awal_out[i];
		host_err[i] = awal_err[i];
	}
	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		cpuJacobi(host_in, host_out, host_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS, totalElemen);
		
		//hitung error
		total_error = 0;
		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		CPUIter++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	CPUTime = t2-t1;

	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "CPUJacobi.hasil", host_out);

	//bebaskan memori
	free(host_in);
	free(host_out);
	free(host_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", CPUIter, CPUTime);
	
	//---CPU PGS---
	printf("\n");
	printf("Metode Point Gauss-Seidel.....");
	//alokasi memori domain
	host_in = (float *)malloc(sizeof(float) * totalElemen);
	host_out = (float *)malloc(sizeof(float) * totalElemen);
	host_err = (float *)malloc(sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	for(int i = 0; i<totalElemen; i++) {
		host_in[i] = awal_in[i];
		host_out[i] = awal_out[i];
		host_err[i] = awal_err[i];
	}
	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		cpuPGS(host_in, host_out, host_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS, totalElemen);
		
		//hitung error
		total_error = 0;
		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		CPUIter2++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	CPUTime2 = t2-t1;

	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "CPUPGS.hasil", host_out);

	//bebaskan memori
	free(host_in);
	free(host_out);
	free(host_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", CPUIter2, CPUTime2);

	//---CPU PSOR---
	printf("\n");
	printf("Metode Point Successive-Over-Relaxation.....");
	//alokasi memori domain
	host_in = (float *)malloc(sizeof(float) * totalElemen);
	host_out = (float *)malloc(sizeof(float) * totalElemen);
	host_err = (float *)malloc(sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	for(int i = 0; i<totalElemen; i++) {
		host_in[i] = awal_in[i];
		host_out[i] = awal_out[i];
		host_err[i] = awal_err[i];
	}
	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		cpuPSOR(host_in, host_out, host_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS, totalElemen, omega);
		
		//hitung error
		total_error = 0;
		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		CPUIter3++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	CPUTime3 = t2-t1;

	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "CPUPSOR.hasil", host_out);

	//bebaskan memori
	free(host_in);
	//free(host_out);
	//free(host_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", CPUIter3, CPUTime3);
	
	printf("\n----\n");

	printf("\nGPU: 1x1");
	//sebelum GPU:inisialisasi dimensi matriks
	int threadx = 1, thready = 1;
	int blockx, blocky;
	blockx = (SISI_MATRIKS/threadx)+1;
	blocky = (SISI_MATRIKS/thready)+1;
	jumlahBlock = dim3(blockx, blocky);
	threadPerBlock = dim3(threadx,thready);


	//---GPU Jacobi---
	printf("\n");
	printf("GPU Jacobi....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuJacobi<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUJacobi1x1.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter, GPUTime);
	
	//---GPU PGS---
	printf("\n");
	printf("GPU PGS....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuPGSRed<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		gpuPGSBlack<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter2++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime2 = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUPGS1x1.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter2, GPUTime2);

	//---GPU PSOR---
	printf("\n");
	printf("GPU PSOR....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuPSORRed<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS, omega);
		gpuPSORBlack<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS, omega);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",GPUIter3,total_error);
		GPUIter3++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime3 = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUPSOR1x1.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter3, GPUTime3);

	printf("\n----\n");

	printf("\nGPU: 2x2");
	//sebelum GPU:inisialisasi dimensi matriks
	threadx = 2;
	thready = 2;
	blockx = (SISI_MATRIKS/threadx)+1;
	blocky = (SISI_MATRIKS/thready)+1;
	jumlahBlock = dim3(blockx, blocky);
	threadPerBlock = dim3(threadx,thready);
	GPUIter = 0; GPUIter2 = 0; GPUIter3 = 0;
	GPUTime = 0; GPUTime2 = 0; GPUTime3 = 0;


	//---GPU Jacobi---
	printf("\n");
	printf("GPU Jacobi....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuJacobi<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUJacobi2x2.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter, GPUTime);
	
	//---GPU PGS---
	printf("\n");
	printf("GPU PGS....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuPGSRed<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		gpuPGSBlack<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter2++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime2 = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUPGS2x2.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter2, GPUTime2);

	//---GPU PSOR---
	printf("\n");
	printf("GPU PSOR....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuPSORRed<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS, omega);
		gpuPSORBlack<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS, omega);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter3++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime3 = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUPSOR2x2.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter3, GPUTime3);

	printf("\n----\n");

	printf("\nGPU: 4x4");
	//sebelum GPU:inisialisasi dimensi matriks
	threadx = 4;
	thready = 4;
	blockx = (SISI_MATRIKS/threadx)+1;
	blocky = (SISI_MATRIKS/thready)+1;
	jumlahBlock = dim3(blockx, blocky);
	threadPerBlock = dim3(threadx,thready);
	GPUIter = 0; GPUIter2 = 0; GPUIter3 = 0;
	GPUTime = 0; GPUTime2 = 0; GPUTime3 = 0;


	//---GPU Jacobi---
	printf("\n");
	printf("GPU Jacobi....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuJacobi<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUJacobi4x4.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter, GPUTime);
	
	//---GPU PGS---
	printf("\n");
	printf("GPU PGS....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuPGSRed<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		gpuPGSBlack<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter2++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime2 = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUPGS4x4.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter2, GPUTime2);

	//---GPU PSOR---
	printf("\n");
	printf("GPU PSOR....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuPSORRed<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS, omega);
		gpuPSORBlack<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS, omega);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter3++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime3 = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUPSOR4x4.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter3, GPUTime3);
	
	printf("\n----\n");

	printf("\nGPU: 8x8");
	//sebelum GPU:inisialisasi dimensi matriks
	threadx = 8;
	thready = 8;
	blockx = (SISI_MATRIKS/threadx)+1;
	blocky = (SISI_MATRIKS/thready)+1;
	jumlahBlock = dim3(blockx, blocky);
	threadPerBlock = dim3(threadx,thready);
	GPUIter = 0; GPUIter2 = 0; GPUIter3 = 0;
	GPUTime = 0; GPUTime2 = 0; GPUTime3 = 0;


	//---GPU Jacobi---
	printf("\n");
	printf("GPU Jacobi....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuJacobi<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUJacobi8x8.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter, GPUTime);
	
	//---GPU PGS---
	printf("\n");
	printf("GPU PGS....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuPGSRed<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		gpuPGSBlack<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter2++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime2 = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUPGS8x8.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter2, GPUTime2);

	//---GPU PSOR---
	printf("\n");
	printf("GPU PSOR....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuPSORRed<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS, omega);
		gpuPSORBlack<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS, omega);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter3++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime3 = t2-t1;
	
	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUPSOR8x8.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter3, GPUTime3);

	printf("\n----\n");

	printf("\nGPU: 16x16");
	//sebelum GPU:inisialisasi dimensi matriks
	threadx = 16;
	thready = 16;
	blockx = (SISI_MATRIKS/threadx)+1;
	blocky = (SISI_MATRIKS/thready)+1;
	jumlahBlock = dim3(blockx, blocky);
	threadPerBlock = dim3(threadx,thready);
	GPUIter = 0; GPUIter2 = 0; GPUIter3 = 0;
	GPUTime = 0; GPUTime2 = 0; GPUTime3 = 0;


	//---GPU Jacobi---
	printf("\n");
	printf("GPU Jacobi....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuJacobi<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUJacobi16x16.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter, GPUTime);
	
	//---GPU PGS---
	printf("\n");
	printf("GPU PGS....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuPGSRed<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		gpuPGSBlack<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter2++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime2 = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUPGS16x16.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter2, GPUTime2);

	//---GPU PSOR---
	printf("\n");
	printf("GPU PSOR....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuPSORRed<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS, omega);
		gpuPSORBlack<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS, omega);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter3++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime3 = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUPSOR16x16.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter3, GPUTime3);

	printf("\n----\n");

	printf("\nGPU: 32x32");
	//sebelum GPU:inisialisasi dimensi matriks
	threadx = 32;
	thready = 32;
	blockx = (SISI_MATRIKS/threadx)+1;
	blocky = (SISI_MATRIKS/thready)+1;
	jumlahBlock = dim3(blockx, blocky);
	threadPerBlock = dim3(threadx,thready);
	GPUIter = 0; GPUIter2 = 0; GPUIter3 = 0;
	GPUTime = 0; GPUTime2 = 0; GPUTime3 = 0;


	//---GPU Jacobi---
	printf("\n");
	printf("GPU Jacobi....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuJacobi<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUJacobi32x32.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter, GPUTime);
	
	//---GPU PGS---
	printf("\n");
	printf("GPU PGS....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuPGSRed<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		gpuPGSBlack<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter2++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime2 = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUPGS32x32.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter2, GPUTime2);

	//---GPU PSOR---
	printf("\n");
	printf("GPU PSOR....");
	//alokasi memori domain
	hipMalloc( (void **)&dev_in, sizeof(float) * totalElemen) ;
	hipMalloc( (void **)&dev_err, sizeof(float) * totalElemen);

	//copy data dari matriks awal ke domain
	hipMemcpy(dev_in, awal_in, totalElemen*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_err, awal_err, totalElemen*sizeof(float), hipMemcpyHostToDevice);

	//mulai komputasi
	t1 = clock() / (CLOCKS_PER_SEC / 1000);
	total_error = -1;
	while ((total_error == -1) || (total_error > MAX_ERROR)) {
		gpuPSORRed<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS, omega);
		gpuPSORBlack<<<jumlahBlock, threadPerBlock>>>(dev_in, dev_err, ad_n, ad_e, ad_s, ad_w, SISI_MATRIKS, omega);
		
		//hitung error
		total_error = 0;
		hipMemcpy(host_err, dev_err, totalElemen*sizeof(float), hipMemcpyDeviceToHost);

		for(int i=0; i<totalElemen; i++) {
			total_error += absol(host_err[i]);
		}
		//printf("%d, %f\n",CPUIter,total_error);
		GPUIter3++;
	}
	t2 = clock() / (CLOCKS_PER_SEC / 1000);
	GPUTime3 = t2-t1;

	hipMemcpy(host_out, dev_in, totalElemen*sizeof(float), hipMemcpyDeviceToHost);
	writeToFile(SISI_MATRIKS, SISI_MATRIKS, totalElemen, "GPUPSOR32x32.hasil", host_out);

	//bebaskan memori
	hipFree(dev_in);
	hipFree(dev_err);
	printf("done\n");
	printf("Banyak iterasi = %d\nWaktu komputasi = %f\n", GPUIter3, GPUTime3);

	getchar();
	return 0;
}
